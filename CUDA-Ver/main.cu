#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <bits/stdc++.h>

using namespace std;

const int numberNodes = 14;

struct Agent{
    int size;
    float fitness;
    int genome[numberNodes];
};

/* Arrange the N elements of ARRAY in random order.
   Only effective if N is much smaller than RAND_MAX;
   if this may not be the case, use a better random
   number generator. */
void shuffle(int *array, size_t n)
{
    if (n > 1)
    {
        size_t i;
        for (i = 0; i < n - 1; i++)
        {
            size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
            int t = array[j];
            array[j] = array[i];
            array[i] = t;
        }
    }
}

Agent RandomAgent(int size){
    int g[size];
    for (int i = 0; i < size ; i++){
        g[i] = i;
    }
    shuffle(g,size);
    struct Agent RAgent{};

    RAgent.size = size;
    RAgent.fitness = 0.0;
    for (int i = 0; i < size ; i++){
        RAgent.genome[i] = g[i];
    }
    return RAgent;
}

__device__ Agent NewAgent (const int genome[]){
    struct Agent nw{};
    nw.size = numberNodes;
    nw.fitness = 0.0;
    for (int i = 0; i < numberNodes ; i++){
        nw.genome[i] = genome[i];
    }
    return nw;
}

__device__ void Mutate(struct Agent *agent, hiprandState state) {
    unsigned int p1 =  hiprand(&state) % numberNodes;
    while (true) {
        unsigned int p2 = hiprand(&state) % numberNodes;
        if (p1 != p2) {
            int temp = agent->genome[p1];
            agent->genome[p1] = agent->genome[p2];
            agent->genome[p2] = temp;
            break;
        }
    }
}

__device__ int Find(const int arr[], int e, unsigned int size){
    for (int i = 0 ; i < size ;i++){
        if (arr[i]== e){
            return e;
        }
    }
    return -1;
}

__device__ void CrossPermutation(int a[], int b[], hiprandState state){
    unsigned int crossPoint = hiprand(&state)%(numberNodes-2)+1;
    unsigned int tempSize = numberNodes-crossPoint-1;
    int tempA[numberNodes];
    int tempB[numberNodes];
    int k = 0;
    for (unsigned int i=crossPoint+1;i<numberNodes;i++){
        tempA[k] = a[i];
        tempB[k] = b[i];
        k++;
    }
    unsigned int idenA = crossPoint + 1;
    unsigned int idenB = crossPoint + 1;
    for (int i = 0;i < numberNodes ; i++) {
        if (Find(tempA, b[i], tempSize) != -1) {
            a[idenA] = b[i];
            idenA++;
        }
    }
    for (int i = 0;i < numberNodes ; i++) {
        if (Find(tempB, a[i], tempSize) != -1) {
            b[idenB] = a[i];
            idenB++;
        }
    }

};

__device__ void FitnessFunction(struct Agent *agent, const float *distance){
    float fitness = 0;
    for (int i = 0; i < numberNodes ; i++){
        if (i < numberNodes - 1 ){
            fitness += distance[agent->genome[i]+agent->genome[i+1]*numberNodes];
        }else{
            fitness += distance[agent->genome[i]+agent->genome[0]*numberNodes];
        }
    }
    agent->fitness = fitness;
}

__device__ Agent GetBest(struct Agent a1,struct Agent a2,struct Agent a3){
    if(a1.fitness < a2.fitness && a1.fitness < a3.fitness){
        return a1;
    }else if (a2.fitness < a3.fitness){
        return a2;
    }else{
        return a3;
    }
}

void PrintAgent(struct Agent agent){
    cout<<agent.fitness<< " ";
    for (int i : agent.genome){
        cout<< i << " ";
    }
    cout << endl;
}

__global__ void EvaluateGen(float *DDistance, struct Agent *DIPopulation, struct Agent *DFPopulation, int popSize, float rate) {
    unsigned int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() , tId, 0, &state);
    if(tId < popSize){
        FitnessFunction(&DIPopulation[tId], DDistance);
        if(hiprand_uniform(&state) < rate){
            unsigned int pair =  hiprand(&state)%popSize;
            int n1[numberNodes];
            int n2[numberNodes];
            for (int i=0;i<numberNodes;i++){
                n1[i] = DIPopulation[pair].genome[i];
                n2[i] = DIPopulation[tId].genome[i];
            }
            CrossPermutation(n1,n2,state);
            struct Agent a1 = NewAgent(n1);
            struct Agent a2 = NewAgent(n2);
            Mutate(&a1, state);
            Mutate(&a2, state);
            FitnessFunction(&a1,DDistance);
            FitnessFunction(&a2,DDistance);
            DFPopulation[tId] = GetBest(a1,a2,DIPopulation[tId]);
        }else{
            DFPopulation[tId] = DIPopulation[tId];
        }
    }

}

void CondensedResult(float current[], float *results, float mean, int popSize, int generation){
    float median = 0, best = 0, worst = 0, stDeviation = 0;
    sort(current,current +popSize);

    best = current[0];
    worst = current[popSize-1];
    if (popSize % 2 == 0){
        median = (current[int(popSize/2)]+current[int((popSize/2)+1)])/2;
    } else{
        median = current[int((popSize/2)+1)];
    }
    for(int i = 0;i<popSize;i++){
        stDeviation += pow(current[i]-mean,2);
    }
    stDeviation /= popSize;
    stDeviation = sqrt(stDeviation);
    results[generation*5] = best;
    results[generation*5+1] = worst;
    results[generation*5+2] = mean;
    results[generation*5+3] = median;
    results[generation*5+4] = stDeviation;
}

extern "C" {
    void evaluateGen(float *distance, float *results, int popSize, int generations, float rate) {
        srand(time(nullptr));
        struct Agent population[popSize];
        for (int i = 0; i< popSize;i++){
            population[i] = RandomAgent(numberNodes);

        }
        unsigned long DDistanceSize = (numberNodes*numberNodes)*sizeof(float);
        float* DDistance;
        hipMalloc((void**)&DDistance,DDistanceSize);
        hipMemcpy(DDistance,distance,DDistanceSize,hipMemcpyHostToDevice);

        unsigned long DPopulationSize = (sizeof(int)+sizeof(float)+(sizeof(int)*numberNodes))*popSize;
        struct Agent* DIPopulation;
        hipMalloc((void**)&DIPopulation,DPopulationSize);

        struct Agent* DFPopulation;
        hipMalloc((void**)&DFPopulation,DPopulationSize);
        for (int i = 0; i < generations;i++){
            float popResults[popSize];
            float mean = 0.0;

            hipMemcpy(DIPopulation,population,DPopulationSize,hipMemcpyHostToDevice);

            EvaluateGen<<<256,int(popSize/256)+1>>>(DDistance, DIPopulation, DFPopulation, popSize, rate);

            hipMemcpy(population,DFPopulation,DPopulationSize,hipMemcpyDeviceToHost);

            for (int j = 0; j< popSize;j++){
                mean += population[j].fitness;
                popResults[j] = population[j].fitness;
            }
            mean /= popSize;
            CondensedResult(popResults,results,mean, popSize,i);
        }
        hipFree(DDistance);
        hipFree(DIPopulation);
        hipFree(DFPopulation);

    }
}


